#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

template <typename T>
struct CudaAllocator{
    using value_type = T;

    T *allocate(size_t size)
    {
        T *ptr = nullptr;
        hipMallocManaged(&ptr, size * sizeof(T));
        return ptr;
    }

    void deallocate(T *ptr, size_t size = 0)
    {
        hipFree(ptr);
    }
};


__device__ __host__ __inline__ void say_hello()
{
#ifdef __CUDA_ARCH__
    printf("device hello cuda %d %d %d\n", __CUDA_ARCH__, threadIdx.x, blockDim.x);
#else
    printf("device hello cpu\n");
#endif
}

__global__ void kernel()
{
    say_hello();
}

template <typename Func>
__global__ void parallel_for(int n, Func func)
{
    for (int i = 0; i < n; i++)
        func(i);
}

int main(int, char**) {
    constexpr int n = 100;
    std::vector<float, CudaAllocator<float>> arr(n);
    parallel_for<<<1, 1>>>(n, [arr = arr.data()] __device__ (int i)
    {
        arr[i] = sinf(i);
    });
    hipDeviceSynchronize();
    for (int i = 0; i < n; i++)
    {
        printf("%f\n", arr[i]);
    }
}
